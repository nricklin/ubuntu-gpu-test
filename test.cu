#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#ifndef USE_MALLOC_HOST
#define USE_MALLOC_HOST
#endif
#undef USE_MALLOC_HOST

#define CUDA_SAFE_CALL(err)  __cudaSafeCall(err,__FILE__,__LINE__)

inline void __cudaSafeCall(hipError_t err,const char *file, const int line) {
  if(hipSuccess != err) {
    printf("%s(%i) : cudaSafeCall() Runtime API error : %s.\n",
           file, line, hipGetErrorString(err) );
    exit(-1);
  }
}

int main(int argc, char * argv[]) {
  int numDevs=0, i, j, N, nBytes;
  hipError_t err;
  hipDeviceProp_t prop;
  hipEvent_t start, stop;
  float *x_cpu, *y_cpu, *x_gpu;
  float dt,totalDtTo=0.0,totalDtFrom=0.0;

  /* create events */
  CUDA_SAFE_CALL(hipEventCreate(&start));
  CUDA_SAFE_CALL(hipEventCreate(&stop));

  N = atoi(argv[1]);
  nBytes = N*sizeof(float);
#ifdef USE_MALLOC_HOST
  CUDA_SAFE_CALL(cudaMallocHost((void**)&x_cpu,nBytes));
  CUDA_SAFE_CALL(cudaMallocHost((void**)&y_cpu,nBytes));
#else
  x_cpu = (float *) malloc(nBytes);
  y_cpu = (float *) malloc(nBytes);
#endif

  for (i=0; i<N; ++i) {
    x_cpu[i] = 1.0*i;
  }

  CUDA_SAFE_CALL(hipGetDeviceCount(&numDevs));
  printf("Number of CUDA Devices = %d\n",numDevs);
  printf("===========================\n");

  for (i=0; i<numDevs; ++i) {
    CUDA_SAFE_CALL(hipSetDevice(i));
    CUDA_SAFE_CALL(hipMalloc((void**)&x_gpu,nBytes));
#ifndef USE_MALLOC_HOST
    CUDA_SAFE_CALL(hipHostRegister(x_cpu, nBytes, hipHostRegisterMapped));
    CUDA_SAFE_CALL(hipHostRegister(y_cpu, nBytes, hipHostRegisterMapped));
#endif

    CUDA_SAFE_CALL(hipGetDeviceProperties(&prop,i));
    printf("Device %d has name %s with compute capability %d.%d canMapHostMemory=%d\n",i,prop.name,prop.major,prop.minor,prop.canMapHostMemory);
    printf("                           global memory = %1.5g\n",1.0*prop.totalGlobalMem/(1024*1024*1024));

    dt=0.0;
    CUDA_SAFE_CALL(hipEventRecord(start, 0));
    for (j=0; j<100; ++j) {
      CUDA_SAFE_CALL(hipMemcpy(x_gpu, x_cpu, nBytes, hipMemcpyHostToDevice));
    }
    CUDA_SAFE_CALL(hipEventRecord(stop, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(stop));
    CUDA_SAFE_CALL(hipEventElapsedTime(&dt,start,stop));
    totalDtTo+=dt;

    dt=0.0;
    CUDA_SAFE_CALL(hipEventRecord(start, 0));
    for (j=0; j<100; ++j) {
      CUDA_SAFE_CALL(hipMemcpy(y_cpu, x_gpu, nBytes, hipMemcpyDeviceToHost));
    }
    CUDA_SAFE_CALL(hipEventRecord(stop, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(stop));
    CUDA_SAFE_CALL(hipEventElapsedTime(&dt,start,stop));
    totalDtFrom+=dt;
    
    totalDtTo*=.001;
    totalDtFrom*=.001;

    printf("HostToDevice PCI Express BW=%g GB/s\n",100.0*nBytes/(1024*1024*1024)/totalDtTo);
    printf("DeviceToHost PCI Express BW=%g GB/s\n",100.0*nBytes/(1024*1024*1024)/totalDtFrom);

    totalDtTo=0.0;
    totalDtFrom=0.0;
#ifndef USE_MALLOC_HOST
    CUDA_SAFE_CALL(hipHostUnregister(x_cpu));
    CUDA_SAFE_CALL(hipHostUnregister(y_cpu));
#endif
    CUDA_SAFE_CALL(hipFree(x_gpu));
  }

#ifndef USE_MALLOC_HOST
  free(x_cpu);
  free(y_cpu);
#else
  CUDA_SAFE_CALL(cudaFreeHost(x_cpu));
  CUDA_SAFE_CALL(cudaFreeHost(y_cpu));
#endif

  /* destroy events */
  CUDA_SAFE_CALL(hipEventDestroy(start));
  CUDA_SAFE_CALL(hipEventDestroy(stop));
  return 0;
}
